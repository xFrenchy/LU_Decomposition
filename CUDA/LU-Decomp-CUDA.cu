#include "hip/hip_runtime.h"
/** @file
  * Name: Parallel LU Decomposition - CUDA Version
  * Authored by: Team Segfault
  * Description: This program performs Lower/Upper decomposition on a square matrix and 
  * subsequently solves the associated system of equations with Forward and Backward substitution.
  * Implementation Date: 11/23/2020
*/

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void DeleteMatrix(float**,int);
void PrintMatrix(float **, int);
void InitializeMatrices(float **&, float **&, float **&, int);
bool GetUserInput(int, char *[], int&,int&);
void sequentialLUdecomposition(float**, float** &, int);


__global__ void RowOperations(float *lower, float *upper, int i, int thicness){

	// Let us get this diagonal thing out of the way
	if(blockIdx.x * blockDim.x  + threadIdx.x == 0) 
		lower[ i*thicness + i ] = 1; 

	int k = blockIdx.x + i + 1;
	int j = threadIdx.x + i;
	
	if( !( k < thicness && j < thicness) ) return; // Whoops

	__shared__ double pivot;

	// And get one pivot per block
	if(threadIdx.x == 0) 
		pivot = -1.0/upper[ i*thicness + i ];
	

	 // Hey guys! Wait up!
	__syncthreads();

	// It is worth noting that the matrices are column major here
	lower[k + thicness*i] = upper[k + thicness*i]/upper[i + thicness*i];
	upper[k + thicness*j] = upper[k + thicness*j] + pivot*upper[k + thicness*i] * upper[i + thicness*j];
	
    
}

void cudaLUDecomp(float *d_lower, float *d_upper, int thicness){

	int i, numBlocks, numThreads;

	for(i = 0; i < thicness; ++i){

		// Since all of these are square these are the same.
		numBlocks = numThreads = thicness-i;

		dim3 dimGrid(numBlocks,1);	
		dim3 dimBlock(numThreads,1);	

		RowOperations<<<dimGrid,dimBlock>>>(d_lower, d_upper, i, thicness);
	}
}


//------------------------------------------------------------------
// Main Program
//------------------------------------------------------------------
int main(int argc, char *argv[]){
	srand(time(NULL));	//set the seed
	
	//Matrices
	float **a, **lower, **upper;
	//Device pointers
	float *d_lower, *d_upper;
	
	int	n,isPrintMatrix;
	float runtime;

	//Get program input
	if (!GetUserInput(argc,argv,n,isPrintMatrix)) return 1;

	//Initialize the matrices
	// a == upper and lower -> 0
	InitializeMatrices(a, lower, upper, n);

	//Get start time
	runtime = clock()/(float)CLOCKS_PER_SEC;

	hipMalloc((void**)&d_lower, n*n*sizeof(float));
	hipMalloc((void**)&d_upper, n*n*sizeof(float));
	hipMemcpy(d_upper, upper[0], n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_lower, lower[0], n*n*sizeof(float), hipMemcpyHostToDevice);

	cudaLUDecomp(d_lower, d_upper, n);

	
    //Get results from the device
    hipMemcpy(lower[0],d_lower, n*n*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(upper[0],d_upper, n*n*sizeof(float),hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	runtime = clock() - runtime; //Make note of LU Decomp


	// TODO: Write the substitution function. That is a future problem.
	
	printf("A:\n");
	PrintMatrix(a,n); 

	printf("Lower:\n");
	PrintMatrix(lower,n); 

	printf("Upper:\n");
	PrintMatrix(upper,n); 

	printf("LU Decomposition ran in %.2f seconds\n", (runtime)/float(CLOCKS_PER_SEC));
	
	hipFree(d_lower);
	hipFree(d_upper);

	DeleteMatrix(upper,n);	
	DeleteMatrix(lower,n);	
	DeleteMatrix(a,n);	

	return 0;
}

void sequentialLUdecomposition(float** a, float** &l, int n)
{
    for (int i = 0; i < n; i++)
    {
        
        float temp;
        float pivot = -1.0/a[i][i];
        
        l[i][i] = 1;
        
        for (int k = i+1; k < n; k++)
        {
            temp = pivot*a[k][i];
            l[k][i] = a[k][i]/a[i][i];
            for (int j = i; j < n; j++)
            {
                a[k][j] = a[k][j] + temp * a[i][j];
            }
        }
        
    }
}



//-----------------------------------------------------------------------
//   Get user input of matrix dimension and printing option
//-----------------------------------------------------------------------
bool GetUserInput(int argc, char *argv[],int& n,int& isPrint)
{
	bool isOK = true;
	
	if(argc < 2) 
	{
		printf("Arguments:<X> [<Y>]");
		printf("X : Matrix size [X x X]");
		printf("Y = 1: print the input/output matrix if X < 10");
		printf("Y <> 1 or missing: does not print the input/output matrix");
		isOK = false;
	}
	else 
	{
		//get matrix size
		n = atoi(argv[1]);
		if (n <=0) 
		{
			printf("Matrix size must be larger than 0");
			isOK = false;
		}
		//is print the input/output matrix
		if (argc >=3)
			isPrint = (atoi(argv[2])==1 && n <=9)?1:0;
		else
			isPrint = 0;
	}
	return isOK;
}


//------------------------------------------------------------------
//delete matrix matrix a[n x n]
//------------------------------------------------------------------
void DeleteMatrix(float **a,int n)
{
	delete[] a[0];
	delete[] a; 
}

//------------------------------------------------------------------------------------------------
//Fills matrix A with random values, upper and lower is filled with 0's except for their diagonals
//------------------------------------------------------------------------------------------------
void InitializeMatrices(float **&a, float **&lower, float **&upper, int size){
	a = new float*[size];
	a[0] = new float[size*size];
	for (int i = 1; i < size; i++)	
		a[i] = a[i-1] + size;
	lower = new float*[size];
	lower[0] = new float[size*size];
	for (int i = 1; i < size; i++)	
		lower[i] = lower[i-1] + size;
	upper = new float*[size];
	upper[0] = new float[size*size];
	for (int i = 1; i < size; i++)	
		upper[i] = upper[i-1] + size;
	
	for(int i = 0; i < size; ++i){
		for(int j = 0; j < size; ++j){
			upper[i][j] = a[i][j] = (rand() % 11) + 1;
			lower[i][j] = 0;
		}
	}
}


//------------------------------------------------------------------
//Print the matrix that was passed to it
//------------------------------------------------------------------
void PrintMatrix(float **matrix, int size) 
{
	for (int i = 0 ; i < size ; i++){
		for (int j = 0 ; j < size ; j++){
			printf("%.2f\t", matrix[j][i]);
		}
		printf("\n");
	}
}